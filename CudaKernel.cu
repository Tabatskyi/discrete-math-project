
#include <hip/hip_runtime.h>
extern "C" __device__ double atomicAddDouble(double* address, double val) {
#if __CUDA_ARCH__ >= 600
	return atomicAdd(address, val);
#else
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
			__double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
#endif
}

extern "C" __global__ void ConstructTours(double* graph, int* tours, double* pheromones, int numAnts, int numCities)
{
    int antIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (antIndex < numAnts)
    {
		int cityIndex = 0;
		tours[antIndex * numCities + cityIndex] = cityIndex;
		for (cityIndex = 1; cityIndex < numCities; cityIndex++)
		{
			int currentCity = tours[antIndex * numCities + cityIndex - 1];
			int nextCity = -1;
			double maxPheromone = 0.0;
			for (int i = 0; i < numCities; i++)
			{
				if (graph[currentCity * numCities + i] > 0)
				{
					int visited = 0;
					for (int j = 0; j < cityIndex; j++)
					{
						if (tours[antIndex * numCities + j] == i)
						{
							visited = 1;
							break;
						}
					}
					if (!visited)
					{
						double pheromone = pheromones[currentCity * numCities + i];
						if (pheromone > maxPheromone)
						{
							maxPheromone = pheromone;
							nextCity = i;
						}
					}
				}
			}

			if (nextCity == -1)
			{
				for (int i = 0; i < numCities; i++)
				{
					int visited = 0;
					for (int j = 0; j < cityIndex; j++)
					{
						if (tours[antIndex * numCities + j] == i)
						{
							visited = 1;
							break;
						}
					}
					if (!visited)
					{
						nextCity = i;
						break;
					}
				}
			}

			tours[antIndex * numCities + cityIndex] = nextCity;
		}
    }
}

extern "C" __global__ void UpdatePheromones(double* pheromones, int* tours, int numAnts, int numCities)
{
    int cityIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (cityIndex < numCities)
    {
		for (int i = 0; i < numCities; i++)
		{
			for (int j = 0; j < numAnts; j++)
			{
				int city1 = tours[j * numCities + cityIndex];
				int city2 = tours[j * numCities + (cityIndex + 1) % numCities];
				if (city1 != -1 && city2 != -1)
				{
					atomicAddDouble(&pheromones[city1 * numCities + city2], 1.0);
					atomicAddDouble(&pheromones[city2 * numCities + city1], 1.0);
				}
			}
		}
    }
}
